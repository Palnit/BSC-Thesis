#include "hip/hip_runtime.h"
//
// Created by Palnit on 2024. 01. 21.
//

#include "Dog/cuda/dog_edge_detector_cuda.cuh"
#include "general/cuda/gauss_blur.cuh"

__global__ void DifferenceOfGaussian(float* kernel1,
                                     float* kernel2,
                                     float* finalKernel,
                                     int kernelSize) {
    uint32_t x = threadIdx.x;
    uint32_t y = threadIdx.y;

    *(finalKernel + x + (y * kernelSize)) =
        *(kernel1 + x + (y * kernelSize)) - *(kernel2 + x + (y * kernelSize));

}

std::shared_ptr<uint8_t> CudaDogDetector::Detect() {

    m_detected =
        static_cast<uint8_t*>(malloc(sizeof(uint8_t) * m_w * m_h * m_stride));
    uint8_t* d_pixel = nullptr;

    hipMalloc((void**) &d_pixel,
               sizeof(uint8_t) * m_w * m_h
                   * m_stride);

    hipMemcpy(
        d_pixel, m_pixels,
        sizeof(uint8_t) * m_w * m_h * m_stride,
        hipMemcpyHostToDevice);

    float* dest1;
    float* dest2;

    float* kernel1;
    float* kernel2;
    float* finalKernel;

    dim3 threads(32, 32);
    dim3 block
        (m_w / threads.x + (m_w % threads.x == 0 ? 0 : 1),
         m_h / threads.y
             + (m_h % threads.y == 0 ? 0 : 1));

    hipMalloc((void**) &kernel1,
               sizeof(float) * m_gaussKernelSize * m_gaussKernelSize);
    hipMalloc((void**) &kernel2,
               sizeof(float) * m_gaussKernelSize * m_gaussKernelSize);
    hipMalloc((void**) &finalKernel,
               sizeof(float) * m_gaussKernelSize * m_gaussKernelSize);
    hipMalloc((void**) &dest1, sizeof(float) * m_w * m_h);
    hipMalloc((void**) &dest2, sizeof(float) * m_w * m_h);

    dim3 gauss(m_gaussKernelSize, m_gaussKernelSize);
    hipEventRecord(m_timers.All_start);
    hipEventRecord(m_timers.GrayScale_start);
    convertToGreyScale<<<block, threads>>>(d_pixel, dest1, m_w, m_h);
    hipEventRecord(m_timers.GrayScale_stop);
    hipEventSynchronize(m_timers.GrayScale_stop);

    hipEventRecord(m_timers.Gauss1Creation_start);
    GetGaussian<<<1, gauss>>>(kernel1, m_gaussKernelSize, m_standardDeviation1);
    hipEventRecord(m_timers.Gauss1Creation_stop);
    hipEventSynchronize(m_timers.Gauss1Creation_stop);

    hipEventRecord(m_timers.Gauss2Creation_start);
    GetGaussian<<<1, gauss>>>(kernel2, m_gaussKernelSize, m_standardDeviation2);
    hipEventRecord(m_timers.Gauss2Creation_stop);
    hipEventSynchronize(m_timers.Gauss2Creation_stop);

    hipEventRecord(m_timers.DifferenceOfGaussian_start);
    DifferenceOfGaussian<<<1, gauss>>>(kernel1,
                                       kernel2,
                                       finalKernel,
                                       m_gaussKernelSize);
    hipEventRecord(m_timers.DifferenceOfGaussian_stop);
    hipEventSynchronize(m_timers.DifferenceOfGaussian_stop);

    hipEventRecord(m_timers.Convolution_start);
    GaussianFilter<<<block, threads>>>(dest1,
                                       dest2,
                                       finalKernel,
                                       m_gaussKernelSize,
                                       m_w,
                                       m_h);
    hipEventRecord(m_timers.Convolution_stop);
    hipEventSynchronize(m_timers.Convolution_stop);

    CopyBack<<<block, threads>>>(d_pixel, dest2, m_w, m_h);
    hipEventRecord(m_timers.All_stop);
    hipEventSynchronize(m_timers.All_stop);

    hipEventElapsedTime(&m_timings.All_ms,
                         m_timers.All_start,
                         m_timers.All_stop);
    hipEventElapsedTime(&m_timings.GrayScale_ms,
                         m_timers.GrayScale_start,
                         m_timers.GrayScale_stop);
    hipEventElapsedTime(&m_timings.Gauss1Creation_ms,
                         m_timers.Gauss1Creation_start,
                         m_timers.Gauss1Creation_stop);
    hipEventElapsedTime(&m_timings.Gauss2Creation_ms,
                         m_timers.Gauss2Creation_start,
                         m_timers.Gauss2Creation_stop);
    hipEventElapsedTime(&m_timings.DifferenceOfGaussian_ms,
                         m_timers.DifferenceOfGaussian_start,
                         m_timers.DifferenceOfGaussian_stop);
    hipEventElapsedTime(&m_timings.Convolution_ms,
                         m_timers.Convolution_start,
                         m_timers.Convolution_stop);

    hipMemcpy(m_detected, d_pixel,
               sizeof(uint8_t) * m_w * m_h * m_stride,
               hipMemcpyDeviceToHost);

    hipFree(dest1);
    hipFree(dest2);
    hipFree(kernel1);
    hipFree(kernel2);
    hipFree(finalKernel);
    hipFree(d_pixel);
    hipDeviceSynchronize();
    return std::shared_ptr<uint8_t>(m_detected);
}
